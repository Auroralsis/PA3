#include "hip/hip_runtime.h"
#include "spmm_opt.h"

const int WARP_SIZE = 32;

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE) {
    // ptr, idx, val分别是稀疏矩阵的CSR格式对应的数组
    // vin, vout分别是与稀疏矩阵相乘的稠密矩阵和最终结果的稠密矩阵
    // num_v是稀疏矩阵的行数，即M*M中的M
    // INFEATURE是输入的稠密矩阵的列数，M*K中的K

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // 根据线程id求出该线程负责的具体稀疏矩阵中的一行以及其需要计算的列
    int row_of_thr = tid / WARP_SIZE;
    int line_of_thr = tid % WARP_SIZE;

    // 对于INFERTURE=32 或 256，计算每个线程应该负责的列的个数
    int lines_num = INFEATURE / WARP_SIZE;

    if (row_of_thr >= num_v) return;
    int begin = ptr[row_of_thr], end = ptr[row_of_thr + 1];

    for (int j = 0; j < lines_num; j++) {
        float result = 0.0f;
        for (int i = begin; i < end; i++) {
            result += vin[idx[i] * INFEATURE + j * 32 + line_of_thr] * val[i];
        }
        vout[row_of_thr * INFEATURE + j * 32 + line_of_thr] = result;
    }
}

void SpMMOpt::preprocess(float *vin, float *vout) {
    // TODO: your code
    const int ROW_SIZE = 1;
    int BLOCK_SIZE = WARP_SIZE * ROW_SIZE;
    grid.x = (num_v + ROW_SIZE - 1) / ROW_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout) {
    // TODO: your code
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}